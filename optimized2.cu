#include "hip/hip_runtime.h"
#include "project.h"
using namespace std;


/*
First optimized GPU version of hough
point points[] = array containing points
long pointcount = number of points
Each thread has one row of accumulator to write to and adds votes to it directly.
Gives local row to avoid global memory accesses, saves A LOT of time.
*/
__global__ void optimizedHough2(unsigned long long* accumulator, int pointCount){

    int index=threadIdx.x+(blockDim.x*blockIdx.x);
    if(index<32400){
        int phi,theta;
        double sinp,cosp,sint,cost;
        double conversion=M_PI/180;
        phi=index%180;
        theta=index/180;
        sinp=sin(phi*conversion);
        cosp=cos(phi*conversion);
        sint=sin(theta*conversion);
        cost=cos(theta*conversion);
        point p;
        double rho;
        ushort localAccumulator[184];
        memset(localAccumulator,0,184*sizeof(ushort));
        for(int i=0;i<pointCount;i++){
                p=c_points[i];
                rho=p.x*sinp*cost
                +p.y*sinp*sint
                +p.z*cosp;
                localAccumulator[((int)floor(rho)+92)]++;
        }
        //Write entire local row to global accumulator at end of thread to decrease global memory accesses.
        for(int r=0;r<184;r++){
            accumulator[phi*(180)*(2*92)+theta*2*92+r]+=localAccumulator[r];
        }

    }
}

//"Main" of second optimized version
unsigned long long* optimized_compute_on_device2(int winnerCount,point* h_points, long pointCount,int rad){
    long bytes=pointCount*sizeof(point);
    long accumBytes=2*92*180*180*sizeof(unsigned long long);
    long winnerBytes=winnerCount*sizeof(unsigned long long);

    //Initialize device input vectors
    point* d_points;


	//Initialize output matrices
    unsigned long long* d_accumulator,*h_accumulator,*d_output, *d_winners, *h_winners;

    //Allocate memory
    h_accumulator=new unsigned long long[92*180*360]();
    h_winners=new unsigned long long[winnerCount]();
    hipMalloc(&d_points, bytes);
    hipMalloc(&d_accumulator,accumBytes);
    hipMalloc(&d_output,accumBytes);
    hipMalloc(&d_winners,winnerBytes);

    //Set accumulator to start at 0;
    hipMemset(&d_accumulator,0,accumBytes);
    







    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    //Run kernel
    int blockSize=1024;
    int gridSize=ceil((double)(180*180)/(double)1024);
    long pointsLeft=pointCount;
    long startIndex=0;
    //int chunkSize=2047;
    // int i;
    // int chunkCount=pointCount/chunkSize;
    // for(i=0;i<chunkCount;i++){
    //     hipMemcpyToSymbol(HIP_SYMBOL(c_points),(&h_points[i*chunkSize]),sizeof(point)*chunkSize,hipMemcpyHostToDevice);
    //     optimizedHough2<<<gridSize,blockSize>>>(d_accumulator,chunkSize);
    // }
    // if(pointCount%chunkSize!=0){
    //     hipMemcpyToSymbol(HIP_SYMBOL(c_points),(&h_points[chunkCount*chunkSize]),sizeof(point)*chunkSize,hipMemcpyHostToDevice);
    //     optimizedHough2<<<gridSize,blockSize>>>(d_accumulator,pointCount-(chunkSize*chunkCount));

    // }
    while(pointsLeft>0){
        if(pointsLeft>=2047){
            pointCount=2047;
            pointsLeft-=2047;
        }
        else{
            pointCount=pointsLeft;
            pointsLeft=0;
        }
        hipMemcpyToSymbol(HIP_SYMBOL(c_points),&h_points[startIndex],sizeof(point)*pointCount);
        optimizedHough2<<<gridSize,blockSize>>>(d_accumulator,pointCount);
        startIndex+=pointCount;
    }
    
    gridSize=ceil((double)(2*92*180*180)/(double)1024);
    naiveSupression<<<gridSize,blockSize>>>(rad,d_accumulator,2*92,180,180,d_output);
    naiveDetermineWinners<<<gridSize,blockSize>>>(winnerCount,d_output,2*92,180,180,d_winners);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf("GPU Version Time elapsed: %f ms\n",time);




	//Copy output to host and deallocate device memory.
    hipMemcpy(h_winners,d_winners,winnerBytes,hipMemcpyDeviceToHost);
    //hipMemcpy(h_accumulator,d_output,accumBytes,hipMemcpyDeviceToHost);
	hipFree(d_points);
    hipFree(d_accumulator);
    hipFree(d_output);
    hipFree(d_winners);
    delete[] h_accumulator;
	return h_winners;
}
